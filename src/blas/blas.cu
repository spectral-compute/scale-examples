
#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>

#include <hipblas.h>


void check(hipError_t error, const char * file, size_t line) {
    if (error != hipSuccess)
    {
        std::cout << "cuda error: " << hipGetErrorString(error) << " at " << file << ":" << line << std::endl;
        exit(1);
    }
}


void checkCublas(hipblasStatus_t error, const char * file, size_t line) {
    if (error != HIPBLAS_STATUS_SUCCESS) {
        std::cout << "cublas error: " << cublasGetStatusString(error) << " at " << file << ":" << line << std::endl;
        exit(1);
    }
}


#define CHECK(error) check(error, __FILE__, __LINE__)
#define CHECK_CUBLAS(error) checkCublas(error, __FILE__, __LINE__)


int main(int argc, char ** argv) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    const size_t N = 10;
    const size_t BYTES = N * sizeof(double);
    const double E = 1e-5;

    /* Prepare the data */

    std::vector<double> A(N);
    std::vector<double> B(N);

    for (size_t i = 0; i < N; i++) {
        A[i] = i;
        B[i] = i + N;
    }

    /* Send the data */

    double * devA;
    double * devB;

    CHECK(hipMalloc(&devA, BYTES));
    CHECK(hipMalloc(&devB, BYTES));

    CHECK(hipMemcpy(devA, A.data(), BYTES, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devB, B.data(), BYTES, hipMemcpyHostToDevice));

    /* Calculate */

    const int strideA = 1;
    const int strideB = 1;
    double result = 0;

    CHECK_CUBLAS(hipblasDdot(handle, A.size(), devA, strideA, devB, strideB, &result));

    CHECK(hipDeviceSynchronize());

    double expected = 0;
    for (size_t i = 0; i < N; i++) {
        expected += A[i] * B[i];
    }

    if (std::abs(result - expected) > E) {
        std::cout << "Result " << result << " is different from expected " << expected << std::endl;
    }

    CHECK_CUBLAS(hipblasDestroy(handle));

    std::cout << "Example finished." << std::endl;

    return 0;
}
