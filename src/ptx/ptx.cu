
#include <hip/hip_runtime.h>
#include <bitset>
#include <vector>
#include <iostream>
#include <cstdint>


__device__ inline uint32_t ptx_add(uint32_t x, uint32_t y) {
    // Calculate a sum of `x` and `y`, put the result into `x`
    asm(
        "add.u32 %0, %0, %1;"
        : "+r"(x)
        : "r"(y)
    );
    return x;
}


__global__ void kernelAdd(const uint32_t * a, const uint32_t * b, size_t n, uint32_t * out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n)
    {
        out[idx] = ptx_add(a[idx], b[idx]);
    }
}


template<uint8_t Op>
__device__ inline uint32_t ptx_lop3(uint32_t x, uint32_t y, uint32_t z) {
    // Compute operator `Op` on `x`, `y`, `z`, put the result into `x`

    asm(
        "lop3.b32 %0, %0, %1, %2, %3;"
        : "+r"(x)
        : "r"(y), "r"(z), "n"(Op)
    );
    return x;
}


template<uint8_t Op>
__global__ void kernelLop3(const uint32_t * a, const uint32_t * b, const uint32_t * c, size_t n, uint32_t * out) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < n)
    {
        out[idx] = ptx_lop3<Op>(a[idx], b[idx], c[idx]);
    }
}


void check(hipError_t error, const char * file, size_t line) {
    if (error != hipSuccess)
    {
        std::cout << "cuda error: " << hipGetErrorString(error) << " at " << file << ":" << line << std::endl;
        exit(1);
    }
}


#define CHECK(error) check(error, __FILE__, __LINE__)


template<typename T>
constexpr T lop3op(T a, T b, T c) {
    return a & b ^ (~c);
}


int main(int argc, char ** argv) {

    const size_t N = 4096;
    const size_t BYTES = N * sizeof(uint32_t);

    std::vector<uint32_t> a(N);
    std::vector<uint32_t> b(N);
    std::vector<uint32_t> c(N);
    std::vector<uint32_t> out(N);

    for (size_t i = 0; i < N; i++) {
        a[i] = i * 2;
        b[i] = N - i;
        c[i] = i * i;
    }

    uint32_t * devA;
    uint32_t * devB;
    uint32_t * devC;
    uint32_t * devOut;

    CHECK(hipMalloc(&devA, BYTES));
    CHECK(hipMalloc(&devB, BYTES));
    CHECK(hipMalloc(&devC, BYTES));
    CHECK(hipMalloc(&devOut, BYTES));

    CHECK(hipMemcpy(devA, a.data(), BYTES, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devB, b.data(), BYTES, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(devC, c.data(), BYTES, hipMemcpyHostToDevice));

    // Test "add"

    kernelAdd<<<N / 256 + 1, 256>>>(devA, devB, N, devOut);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(out.data(), devOut, BYTES, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < N; i++) {
        if (a[i] + b[i] != out[i]) {
            std::cout << "Incorrect add: " << a[i] << " + " << b[i] << " = " << out[i] << " ?\n";
        }
    }

    // Test "lop3"

    constexpr uint8_t TA = 0xF0;
    constexpr uint8_t TB = 0xCC;
    constexpr uint8_t TC = 0xAA;
    constexpr uint8_t Op = lop3op(TA, TB, TC);

    kernelLop3<Op><<<N / 256 + 1, 256>>>(devA, devB, devC, N, devOut);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());

    CHECK(hipMemcpy(out.data(), devOut, BYTES, hipMemcpyDeviceToHost));

    for (size_t i = 0; i < N; i++) {
        if (lop3op(a[i], b[i], c[i]) != out[i]) {
            std::cout << "Incorrect lop3: \n"
                << "    " << std::bitset<32>{a[i]} << "\n"
                << " &  " << std::bitset<32>{b[i]} << "\n"
                << " ^ ~" << std::bitset<32>{c[i]} << "\n"
                << " =  " << std::bitset<32>{out[i]} << " ?\n\n";
        }
    }

    CHECK(hipFree(devA));
    CHECK(hipFree(devB));
    CHECK(hipFree(devC));
    CHECK(hipFree(devOut));

    // Finish

    std::cout << "Example finished" << std::endl;

    return 0;
}
